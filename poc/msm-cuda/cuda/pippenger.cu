// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <hip/hip_runtime.h>

#include <ff/alt_bn128.hpp>

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

typedef jacobian_t<fp_t> point_t;
typedef xyzz_t<fp_t> bucket_t;
typedef bucket_t::affine_t affine_t;
typedef fr_t scalar_t;

#include <msm/pippenger.cuh>

#ifndef __CUDA_ARCH__

void DumpHex(const void* data, size_t size);

extern "C"
RustError mult_pippenger(point_t* out, const affine_t points[], size_t npoints,
                                       const scalar_t scalars[])
{    
    DumpHex((const void *) &scalars[0], 32);
    DumpHex((const void *) &scalars[1], 32);
    DumpHex((const void *) &scalars[2], 32);
    printf("\n");
    DumpHex((const void *) &points[0], 64);
    printf("\n");
    DumpHex((const void *) &points[1], 64);
    printf("\n");
    DumpHex((const void *) &points[2], 64);
    printf("\n");
    return mult_pippenger<bucket_t>(out, points, npoints, scalars, false);
}

void DumpHex(const void* data, size_t size) {
	char ascii[17];
	size_t i, j;
	ascii[16] = '\0';
	for (i = 0; i < size; ++i) {
		printf("%02X ", ((unsigned char*)data)[i]);
		if (((unsigned char*)data)[i] >= ' ' && ((unsigned char*)data)[i] <= '~') {
			ascii[i % 16] = ((unsigned char*)data)[i];
		} else {
			ascii[i % 16] = '.';
		}
		if ((i+1) % 8 == 0 || i+1 == size) {
			printf(" ");
			if ((i+1) % 16 == 0) {
				printf("|  %s \n", ascii);
			} else if (i+1 == size) {
				ascii[(i+1) % 16] = '\0';
				if ((i+1) % 16 <= 8) {
					printf(" ");
				}
				for (j = (i+1) % 16; j < 16; ++j) {
					printf("   ");
				}
				printf("|  %s \n", ascii);
			}
		}
	}
}
#endif
